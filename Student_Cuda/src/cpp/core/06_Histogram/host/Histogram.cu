#include "hip/hip_runtime.h"
#include <iostream>

#include "Device.h"
#include "Histogram.h"

using std::cout;
using std::endl;

const int SIZE_TAB_SM = 256;

extern __global__ void histogram(int* ptrTabData, int tabSize, int *ptrDevResult);

Histogram::Histogram(int tabSize) :
		tabSize(tabSize)
{
	this->ptrTabResult = new int[SIZE_TAB_SM];
	this->ptrTabData = new int[tabSize];

	// Create data array
	for (int i = 0; i < tabSize; i++)
		ptrTabData[i] = i % SIZE_TAB_SM;

	// Shuffle tab
	for (int i = 0; i < tabSize; i++)
	{
		int a = rand() % tabSize;
		int b = rand() % tabSize;

		// Le meilleur swap du monde
		ptrTabData[a] = ptrTabData[a] ^ ptrTabData[b];
		ptrTabData[b] = ptrTabData[b] ^ ptrTabData[a];
		ptrTabData[a] = ptrTabData[a] ^ ptrTabData[b];
	}

	this->sizeOctetResult = sizeof(int) * SIZE_TAB_SM; // octet
	this->sizeOctetData = sizeof(int) * tabSize;

	// Allocation-party
	HANDLE_ERROR(hipMalloc(&ptrTabOut, sizeOctetResult));
	HANDLE_ERROR(hipMalloc(&ptrTabIn, sizeOctetData));

	// Memory set party
	HANDLE_ERROR(hipMemset(ptrTabOut, 0, sizeOctetResult));
	HANDLE_ERROR(
			hipMemcpy(ptrTabIn, ptrTabData, sizeOctetData,
					hipMemcpyHostToDevice));

	Device::lastCudaError("histogramme.cu constructor"); // temp debug

	// Grid

	this->dg = dim3(16, 2, 1);
	this->db = dim3(256, 1, 1); 	// Produit doit être ^2

	Device::gridHeuristic(dg, db); // optionnel

}

Histogram::~Histogram(void)
{
	Device::lastCudaError("histogram.cu destructor begin"); // temp debug
	HANDLE_ERROR(hipFree(ptrTabOut));
	Device::lastCudaError("histogram.cu destructor end"); // temp debug
}

int* Histogram::run()
{
	Device::lastCudaError("histogram run begin");
	histogram<<<dg,db,sizeOctetResult>>>(ptrTabIn, tabSize, ptrTabOut); // asynchrone
	Device::lastCudaError("histogram run end");

	HANDLE_ERROR(hipMemcpy(ptrTabResult, ptrTabOut, sizeOctetResult ,hipMemcpyDeviceToHost));

	return ptrTabResult;
}
