#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "ReduceTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarlo(hiprandState* ptrTabDevGeneratorGM, int* ptrDevN0, int nbFlechettes);
__global__ void setup_kernel_rand(hiprandState* tabGeneratorThread, int deviceId);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reduceIntraThread(int* tabSM, hiprandState* ptrTabDevGeneratorGM, float a, float b, float M, int nbFlechettes);
static __device__ float f(float x);
static __device__ float toAB(float x, float a, float b);
/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void montecarlo(hiprandState* ptrTabDevGeneratorGM, int* ptrDevN0, float a, float b, float M, int nbFlechettes)
{
	extern __shared__ int tabSM[];
	reduceIntraThread(tabSM, ptrTabDevGeneratorGM, a, b, M, nbFlechettes);
	__syncthreads();
	reduceIntraBlock (tabSM);
	reduceInterBlock(tabSM, ptrDevN0);
}

__global__ void setup_kernel_rand(hiprandState* tabGeneratorThread, int deviceId)
{
	int tid = Indice2D::tid();
	//Customisation du generator: Proposition (au lecteur de faire mieux)
	// Contrainte : Doit etre différent d'un GPU à l'autre
	int deltaSeed = deviceId * INT_MAX;
	int deltaSequence = deviceId * 100;
	int deltaOffset = deviceId * 100;
	int seed = 1234 + deltaSeed;
	int sequenceNumber = tid + deltaSequence;
	int offset = deltaOffset;
	hiprand_init(seed, sequenceNumber, offset, &tabGeneratorThread[tid]);
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void reduceIntraThread(int* tabSM, hiprandState* ptrTabDevGeneratorGM, float a, float b, float M, int nbFlechettes)
{
	const int NB_THREAD = Indice2D::nbThread();
	const int TID = Indice2D::tid();
	const int TID_LOCAL = Indice2D::tidLocal();

	int s = TID;

	int sumThread = 0;
	hiprandState localState = ptrTabDevGeneratorGM[TID];

	while (s < nbFlechettes)
	{
		float x = toAB(hiprand_uniform(&localState), a, b);
		float y = toAB(hiprand_uniform(&localState), 0.0f, M);
		if(y < f(x))
		{
			sumThread++;
		}
		s += NB_THREAD;
	}
	tabSM[TID_LOCAL] = sumThread;
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
__device__ float toAB(float x, float a, float b)
{
	return (b - a) / 1.0f * x + a;
}

__device__ float f(float x)
{
	return sqrtf(1.0f - x * x);
}
