
#include <hip/hip_runtime.h>
//#include <iostream>
//#include <assert.h>
//
//#include "Device.h"
//#include "RayTracing.h"
//#include "Sphere.h"
//#include "cudaTools.h"
//
//#include <limits>
//
//using std::cout;
//using std::endl;
//
///* ========== DECLARATION ========== */
//
//extern __global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, float t, uint nbSphere);
//
///* ---------- PUBLIC ---------- */
//
//RayTracing::RayTracing(const Grid &grid, uint width, uint height, float dt, uint nbSphere) :
//		Animable_I<uchar4>(grid, width, height, "RayTracing Roulin")
//{
//	// time
//	this->t = 0;
//	this->dt = dt;
//
//	// Inputs
//	this->nbSphere = nbSphere;
//	this->spheres = new Sphere[nbSphere];
//
//	// Init spheres
//	float margin = 200.f;
//	for(int i = 0; i < this->nbSphere; i++)
//	{
//		float3 center;
//		center.x = randomFloat(margin, width - margin);
//		center.y = randomFloat(margin, height - margin);
//		center.z  = randomFloat(10.f, 2.f * width);
//
//		float radius = randomFloat(20.f, w / 10.f);
//		float hue = randomFloat(0.f, 1.f);
//
//		this->spheres[i] = Sphere(center, radius, hue);
//	}
//}
//
//RayTracing::~RayTracing(void)
//{
//	delete[] spheres;
//}
//
///* ~~~~~~~~~~  METHODS  ~~~~~~~~~~ */
///**
// * Override
// * Call periodicly by the API
// *
// * Note : domaineMath pas use car pas zoomable
// */
//void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
//{
//	Device::lastCudaError("raytracing rgba uchar4 (before)"); // facultatif, for debug only, remove for release
//
//	// start kernel
//	rayTracing<<<dg, db>>>(ptrDevPixels, w, h, t, nbSphere);
//
//	Device::lastCudaError("raytracing rgba uchar4 (after)"); // facultatif, for debug only, remove for release
//}
//
///* ~~~~~~~~~~ OVERRIDES ~~~~~~~~~~ */
//
//void RayTracing::animationStep()
//{
//	this->t += dt;
//}
//
//
//float RayTracing::randomFloat(float min, float max)
//{
//    float random = ((float) rand()) / (float) RAND_MAX;
//    float diff = max - min;
//    float r = random * diff;
//    return min + r;
//}
///**
// * Override (code naturel omp)
// */
