
#include <hip/hip_runtime.h>
//#include "Indice2D.h"
//#include "cudaTools.h"
//#include "Device.h"
//#include "IndiceTools_GPU.h"
//#include "RayTracingMath.h"
//using namespace gpu;
//
//// Attention : 	Choix du nom est impotant!
////		VagueDevice.cu et non Vague.cu
//// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
////		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!
//
///*----------------------------------------------------------------------*\
// |*			Declaration 					*|
// \*---------------------------------------------------------------------*/
//
///*--------------------------------------*\
// |*		Imported	 	*|
// \*-------------------------------------*/
//
///*--------------------------------------*\
// |*		Public			*|
// \*-------------------------------------*/
//
//__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, float t);
//
///*--------------------------------------*\
// |*		Private			*|
// \*-------------------------------------*/
//
///*----------------------------------------------------------------------*\
// |*			Implementation 					*|
// \*---------------------------------------------------------------------*/
//
///*--------------------------------------*\
// |*		Public			*|
// \*-------------------------------------*/
//
//__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, float t)
//{
//	RayTracingMath rayTracingMath = RayTracingMath();
//
//	const int TID = Indice2D::tid();
//	const int NB_THREAD = Indice2D::nbThread();
//	const int WH = w * h;
//
//	uchar4 color;
//	int i, j;
//
//	int s = TID;
//	while (s < WH)
//	{
//		IndiceTools::toIJ(s, w, &i, &j);
//		rayTracingMath.colorXY(&color, i, j, t);
//		ptrDevPixels[s] = color;
//		s += NB_THREAD;
//	}
//}
//
///*--------------------------------------*\
// |*		Private			*|
// \*-------------------------------------*/
//
///*----------------------------------------------------------------------*\
// |*			End	 					*|
// \*---------------------------------------------------------------------*/
//
