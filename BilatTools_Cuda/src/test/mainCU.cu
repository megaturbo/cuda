#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Importation 					*|
 \*---------------------------------------------------------------------*/

#include "cu.h"
#include "cu_cpp.h"

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
  |*		Imported		*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

bool mainCU(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static void lauchKernel(void);
static __global__ void kernelTest(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

bool mainCU(void)
    {
    cout << "[BilatTools_Cuda] : just inlcude to force compilation in .cu" << endl;

    lauchKernel();

    return true;
    }

void lauchKernel(void)
    {
    dim3 dg = dim3(1, 1, 1);
    dim3 db = dim3(1, 1, 1);

    kernelTest<<<dg,db>>>();
    Device::synchronize();
    }

__global__ void kernelTest(void)
    {
    printf("\nHello from kernel\n"); // TODO use classe
    }


/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

