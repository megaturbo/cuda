#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "DamierHueFloatMath.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void damierHueFloat(float* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void damierHueFloat(float* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t)
    {
    DamierHueFloatMath damierHueFloatMath = DamierHueFloatMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    float color;

    double x;
    double y;

    int pixelI; // in [0,h[
    int pixelJ; // in [0,w[

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)

	damierHueFloatMath.colorXY(&color, x, y, t); // update color

	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

