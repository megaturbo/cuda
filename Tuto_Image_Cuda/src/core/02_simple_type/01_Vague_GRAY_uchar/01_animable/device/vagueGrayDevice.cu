#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "VagueGrayMath.h"
using namespace gpu;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vagueGray(uchar* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vagueGray(uchar* ptrDevPixels, uint w, uint h, float t)
    {
    VagueGrayMath vagueGrayMath = VagueGrayMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH=w*h;

    uchar color;
    int pixelI; // in [0,h[
    int pixelJ; // in [0,w[

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); 	// update (pixelI, pixelJ)

	vagueGrayMath.colorIJ(&color,pixelI, pixelJ, t); 	// update color
	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

