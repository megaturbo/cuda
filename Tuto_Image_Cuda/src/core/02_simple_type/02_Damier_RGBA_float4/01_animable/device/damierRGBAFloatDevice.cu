#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "DamierRGBAFloatMath.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void damierRGBAFloat(float4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void damierRGBAFloat(float4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t)
    {
    DamierRGBAFloatMath damierRGBAFloatMath = DamierRGBAFloatMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    float4 color;

    double x;
    double y;

    int pixelI; // in [0,h[
    int pixelJ; // in [0,w[

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)

	damierRGBAFloatMath.colorXY(&color, x, y, t); // update color

	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

